#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

using namespace std;
using namespace cv;

static const int MASK_SIZE = 5;
static const int mask[MASK_SIZE][MASK_SIZE] = {
        {0,1,2,1,0},
        {1,4,8,4,1},
        {2,8,16,8,2},
        {1,4,8,4,1},
        {0,1,2,1,0}
};

static void HandleError(hipError_t err, string file, int line)
{
    if (err != hipSuccess)
    {
        cout << hipGetErrorString(err) << " in " << file << " at line " << line << endl;
        exit(EXIT_FAILURE);
    }
}

__constant__ int dev_mask[MASK_SIZE][MASK_SIZE];
__constant__ int dev_weight;

__global__ void gaussianBlur(uchar * inputImage, uchar * outputImage, long width, long height)
{
    long x = (blockIdx.x * blockDim.x) + threadIdx.x;
    long y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < width-2 && y < height-2 && x>1 && y>1)
    {
        long r=0, g=0, b=0;
        long pixelIn, pixelOut;

        for (int y_m = 0; y_m<5; y_m++)
        {
            for (int x_m = 0; x_m<5; x_m++)
            {
                pixelIn = width*(y + y_m - 2) * 3 + (x + x_m - 2) * 3;

                r += inputImage[pixelIn + 2] * dev_mask[x_m][y_m];
                g += inputImage[pixelIn + 1] * dev_mask[x_m][y_m];
                b += inputImage[pixelIn] * dev_mask[x_m][y_m];
            }
        }
        pixelOut = (width - 4)*(y - 2) * 3 + (x - 2) * 3;
        outputImage[pixelOut + 2] = r / dev_weight;
        outputImage[pixelOut + 1] = g / dev_weight;
        outputImage[pixelOut] = b / dev_weight;
    }
}

void compute(int blockSize, Mat inputImage, string inputName, string outputName) {
    Mat outputImage;
    outputImage = Mat(inputImage.rows, inputImage.cols, CV_8UC3);
    copyMakeBorder(inputImage, inputImage, 2, 2, 2, 2, BORDER_REPLICATE);

    int gridWidth, gridHeight;
    gridWidth = inputImage.cols / blockSize + ((inputImage.cols % blockSize) == 0 ? 0 : 1);
    gridHeight = inputImage.rows / blockSize + ((inputImage.rows % blockSize) == 0 ? 0 : 1);

    long inputSize = sizeof(uchar) * inputImage.rows* inputImage.cols * 3;
    long outputSize = sizeof(uchar) * outputImage.rows* outputImage.cols * 3;

    uchar * dev_inputImage;
    uchar * dev_outputImage;
    HANDLE_ERROR(hipMalloc((void**)& dev_inputImage, inputSize));
    HANDLE_ERROR(hipMalloc((void**)& dev_outputImage, outputSize));

    HANDLE_ERROR(hipMemcpy(dev_inputImage, inputImage.data, inputSize, hipMemcpyHostToDevice));

    dim3 gridSize(gridWidth, gridHeight);
    dim3 threadsPerBlock(blockSize, blockSize);

    hipEvent_t begin, end;
    float time;
    HANDLE_ERROR(hipEventCreate(&begin));
    HANDLE_ERROR(hipEventCreate(&end));
    HANDLE_ERROR(hipEventRecord(begin, 0));

    gaussianBlur <<< gridSize,threadsPerBlock >>> (dev_inputImage, dev_outputImage, inputImage.cols, inputImage.rows);

    HANDLE_ERROR(hipEventRecord(end, 0));
    HANDLE_ERROR(hipEventSynchronize(end));
    HANDLE_ERROR(hipEventElapsedTime(&time, begin, end));

    HANDLE_ERROR(hipMemcpy(outputImage.data, dev_outputImage, outputSize, hipMemcpyDeviceToHost));

    try {
        imwrite(outputName, outputImage);
    }
    catch (Exception &e) {
        cout << "Exception while writing to file " << e.msg;
    }

    HANDLE_ERROR(hipEventDestroy(begin));
    HANDLE_ERROR(hipEventDestroy(end));

    hipFree(dev_inputImage);
    hipFree(dev_outputImage);

//    cout << "X: "<< gridWidth << ", Y: " << gridHeight << endl;
//    cout << "Threads per block: " << blockSize << endl;
    cout << "Czas: " << time << "ms" << endl;
}

int main(int argc, char** argv) {
    Mat inputImage;
    string inputName, outputName;
    int maxThreads = 32;
    int blockSize = 32;
    int weight = 0;

    if (argc < 3)
    {
        cout << "Invalid arugments";
        return -1;
    }
    else
    {
        inputName = argv[1];
        outputName = argv[2];
        if (argc == 4)
        {
            blockSize = atoi(argv[3]);
        }
    }

    inputImage = imread(inputName, CV_LOAD_IMAGE_COLOR);
    if (inputImage.data == false)
    {
        cout << "No image defined";
        return -1;
    }

    for (int xMask = 0; xMask<5; xMask++) {
        for (int yMask = 0; yMask<5; yMask++) {
            weight += mask[xMask][yMask];
        }
    }

    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(dev_mask), &mask, sizeof(int) * MASK_SIZE * MASK_SIZE));
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(dev_weight), &weight, sizeof(int)));

    struct hipDeviceProp_t properties;
    HANDLE_ERROR(hipGetDeviceProperties(&properties, 0));

    if (blockSize <= maxThreads && blockSize > 0)
    {
        compute(blockSize, inputImage, inputName, outputName);
    }
    else
    {
        for(int size=1; size <= maxThreads; size++)
        {
            compute(size, inputImage, inputName, outputName);
        }
    }

    return 0;
}
